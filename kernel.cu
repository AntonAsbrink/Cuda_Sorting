#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <string>
#include <random>
#include <iostream>
#include <fstream>
#include <algorithm>


__global__ void oddEvenGPU(int *arr, int arrSize)
{
    int i = (blockIdx.x*blockDim.x + threadIdx.x) * 2;

	if (i < arrSize) {
		for (int j = 0; j < arrSize/2; j++) {
			for (int k = i; k < arrSize; k += blockDim.x) {
				if (k + 1 < arrSize) {
					if (arr[k] > arr[k + 1]) {
						int temp = arr[k];
						arr[k] = arr[k + 1];
						arr[k + 1] = temp;
					}
				}
			}
			__syncthreads();
			i++;
			for (int k = i; k < arrSize; k += blockDim.x) {
				if (k + 1 < arrSize) {
					if (arr[k] > arr[k + 1]) {
						int temp = arr[k];
						arr[k] = arr[k + 1];
						arr[k + 1] = temp;
					}
				}
			}
			__syncthreads();
			i--;
		}
	}
}

__global__ void oddEvenBlockGPU(int *arr, int arrSize, int oddEven)
{
	int i = (blockIdx.x*blockDim.x + threadIdx.x) * 2 + oddEven; //oddEven should be a 0 or a 1 depending if its running even or odd phase

	int gridSize = gridDim.x * gridDim.y * gridDim.z;

	if (i < arrSize) {
		for (int k = i; k < arrSize; k += gridSize * blockDim.x) {
			if (k + 1 < arrSize) {
				if (arr[k] > arr[k + 1]) {
					int temp = arr[k];
					arr[k] = arr[k + 1];
					arr[k + 1] = temp;
				}
			}
		}
	}
}

__host__
void oddEvenCPU(int * arr, int arrSize) {
	for (int i = 0; i < arrSize; i++) {
		for (int j = i % 2; j < arrSize; j+=2) {
			if (j + 1 < arrSize) {

				if (arr[j] > arr[j + 1]) {
					int temp = arr[j];
					arr[j] = arr[j + 1];
					arr[j + 1] = temp;
				}
			}
		}
	}
}

__host__
bool isSorted(int *arr, int arrSize) {
	for (int i = 0; i < arrSize-1; i++) {
		if (arr[i] > arr[i + 1]) {
			return false;
		}
	}
	return true;
}

__host__
int main(int argc, char *argv[])
{
	//Generate array values
	srand(time(NULL));

	int arrSize = std::stoi(std::string(argv[1]));

	int *arrCPU = new int[arrSize];
	int *arrGPU;
	for (int i = 0; i < arrSize; i++) {
		arrCPU[i] = rand() % 1000;
	}


	//CPU
	if (std::strcmp(argv[2], "cpu") == 0) {
		oddEvenCPU(arrCPU, arrSize);

		if (std::strcmp(argv[5], "output") == 0) {
			std::ofstream file;
			file.open("output.txt");
			file << "CPU is sorted?: " << (isSorted(arrCPU, arrSize) ? "True" : "False") << std::endl;
			for (int i = 0; i < arrSize; i++) {
				file << arrCPU[i] << " ";
			}
		}
	}

	//GPU pararellism test
	if (std::strcmp(argv[2], "gpu") == 0) {
		//CUDA init
		hipError_t cudasStatus = hipSetDevice(1);
		if (hipMalloc(&arrGPU, arrSize * sizeof(int)) == hipErrorOutOfMemory)
			std::cout << "CUDA memory allow failed" << std::endl;
		hipMemcpy(arrGPU, arrCPU, arrSize * sizeof(int), hipMemcpyHostToDevice);

		if (std::strcmp(argv[4], "0") == 0) {
			oddEvenGPU << <1, std::stoi(std::string(argv[3])) >> > (arrGPU, arrSize);
		}
		else {
			for (int i = 0; i < arrSize; i++) {
				oddEvenBlockGPU << < std::stoi(std::string(argv[4])), std::stoi(std::string(argv[3])) >> > (arrGPU, arrSize, i % 2);
			}
		}

		int *g = new int[arrSize];
		hipMemcpy(g, arrGPU, arrSize * sizeof(int), hipMemcpyDeviceToHost);

		if (std::strcmp(argv[5], "output") == 0) {
			std::ofstream file;
			file.open("output.txt");
			file << "GPU is sorted?: " << (isSorted(g, arrSize) ? "True" : "False") << std::endl;
			for (int i = 0; i < arrSize; i++) {
				file << g[i] << " ";
			}
			delete(g);
		}
	}
	

	hipFree(arrGPU);
	delete(arrCPU);

    return 0;
}